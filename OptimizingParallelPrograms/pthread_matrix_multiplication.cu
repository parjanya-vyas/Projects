#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdint.h>
#include <windows.h>
#include <pthread.h>

int MATRIX_SIZE;
int *mat1, *mat2, *mat3;

LARGE_INTEGER start, end;
LARGE_INTEGER freq;

void* pthread_mat_mul(void *arg)
{
	int i, j, k;
	i = *(int *)arg;
	j = *((int *)(arg) + 1);
	for (k = 0; k < MATRIX_SIZE; k++)
	{
		mat3[i * MATRIX_SIZE + j] += mat1[i * MATRIX_SIZE + k] * mat2[k * MATRIX_SIZE + j];
	}

	return (void *)0;
}

void initialize_timing_and_randomness()
{
	srand(time(NULL));
	if (QueryPerformanceFrequency(&freq) == 0)
	{
		printf("Your machine not support high resolution performance counter\n");
		return;
	}
}

void display_matrix(int *mat, int sz)
{
	int i, j;
	for (i = 0; i < sz; i++)
	{
		for (j = 0; j < sz; j++)
		{
			printf("%d\t", mat[i * sz + j]);
		}
		printf("\n");
	}
}

int main()
{
	printf("Enter matrix size:");
	int mat_sz;
	scanf("%d", &mat_sz);
	getchar();

	printf("Would you like to see the random matrices and the resultant matrix?(YES - 1, NO - 0):");
	int is_debug_mode;
	scanf("%d", &is_debug_mode);
	getchar();

	MATRIX_SIZE = mat_sz;
	const int arg_sz = MATRIX_SIZE * MATRIX_SIZE;
	int i = 0, j = 0, k = 0;

	initialize_timing_and_randomness();

	mat1 = (int *)malloc(MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
	for (i = 0; i < MATRIX_SIZE; i++)
	{
		for (j = 0; j < MATRIX_SIZE; j++)
		{
			mat1[i * MATRIX_SIZE + j] = rand() % 10;
		}
	}

	mat2 = (int *)malloc(MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
	for (i = 0; i < MATRIX_SIZE; i++)
	{
		for (j = 0; j < MATRIX_SIZE; j++)
		{
			mat2[i * MATRIX_SIZE + j] = rand() % 10;
		}
	}

	mat3 = (int *)malloc(MATRIX_SIZE * MATRIX_SIZE * sizeof(int *));
	for (i = 0; i < MATRIX_SIZE; i++)
	{
		for (j = 0; j < MATRIX_SIZE; j++)
		{
			mat3[i * MATRIX_SIZE + j] = 0;
		}
	}

	QueryPerformanceCounter(&start);

	pthread_t *threads = (pthread_t *)malloc(MATRIX_SIZE * MATRIX_SIZE * sizeof(pthread_t));
	int *thread_args = (int *)malloc(arg_sz * 2 * sizeof(int));

	for (i = 0; i < MATRIX_SIZE; i++)
	{
		for (j = 0; j < MATRIX_SIZE; j++, k+=2)
		{
			thread_args[k] = i;
			thread_args[k + 1] = j;
			pthread_create(&threads[i * MATRIX_SIZE + j], NULL, pthread_mat_mul, &thread_args[k]);
		}
	}

	for (i = 0; i < MATRIX_SIZE; i++)
	{
		for (j = 0; j < MATRIX_SIZE; j++)
		{
			pthread_join(threads[i * MATRIX_SIZE + j], NULL);
		}
	}

	QueryPerformanceCounter(&end);

	if (is_debug_mode)
	{
		printf("Matrix 1:\n");
		display_matrix(mat1, MATRIX_SIZE);
		printf("Matrix 2:\n");
		display_matrix(mat2, MATRIX_SIZE);
		printf("Result:\n");
		display_matrix(mat3, MATRIX_SIZE);
	}

	printf("Time taken:\n");
	long long time_taken = (end.QuadPart - start.QuadPart) / (freq.QuadPart / 1000);
	printf("%lld ms", time_taken);

	getchar();
	return 0;
}