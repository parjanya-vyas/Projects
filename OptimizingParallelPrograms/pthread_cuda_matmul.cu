#include "hip/hip_runtime.h"
#include <stdio.h>
#include <time.h>
#include <stdint.h>
#include <windows.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""

int NUMBER_OF_SM, NUMBER_OF_THREADS_PER_SM;

LARGE_INTEGER start, end;
LARGE_INTEGER freq;

__global__ void matmul(int *device_mat1, int *device_mat2, int *device_mat3, int sz, int max_thread)
{
	int row, col, ele, i;
	int temp = 0;
	ele = blockIdx.x * max_thread + threadIdx.x + 1;
	row = (ele - 1) / sz;
	if (row >= sz)
		return;
	col = (ele - 1) % sz;
	for (i = 0; i < sz; i++)
	{
		temp += device_mat1[row * sz + i] * device_mat2[i * sz + col];
	}
	device_mat3[row * sz + col] = temp;
}

inline int _ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct
	{
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] =
	{
		{ 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
		{ 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
		{ 0x30, 192 }, // Kepler Generation (SM 3.0) GK10x class
		{ 0x32, 192 }, // Kepler Generation (SM 3.2) GK10x class
		{ 0x35, 192 }, // Kepler Generation (SM 3.5) GK11x class
		{ 0x37, 192 }, // Kepler Generation (SM 3.7) GK21x class
		{ 0x50, 128 }, // Maxwell Generation (SM 5.0) GM10x class
		{ 0x52, 128 }, // Maxwell Generation (SM 5.2) GM20x class
		{ 0x53, 128 }, // Maxwell Generation (SM 5.3) GM20x class
		{ 0x60, 64 }, // Pascal Generation (SM 6.0) GP100 class
		{ 0x61, 128 }, // Pascal Generation (SM 6.1) GP10x class
		{ 0x62, 128 }, // Pascal Generation (SM 6.2) GP10x class
		{ -1, -1 }
	};

	int index = 0;

	while (nGpuArchCoresPerSM[index].SM != -1)
	{
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
		{
			return nGpuArchCoresPerSM[index].Cores;
		}

		index++;
	}

	// If we don't find the values, we default use the previous one to run properly
	printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[index - 1].Cores);
	return nGpuArchCoresPerSM[index - 1].Cores;
}

void initialize_gpu_parameters(hipDeviceProp_t devProp)
{
	NUMBER_OF_SM = devProp.multiProcessorCount;
	NUMBER_OF_THREADS_PER_SM = _ConvertSMVer2Cores(devProp.major, devProp.minor);
}

void initialize_timing_and_randomness()
{
	srand(time(NULL));
	if (QueryPerformanceFrequency(&freq) == 0)
	{
		printf("Your machine not support high resolution performance counter\n");
		return;
	}
}

void display_matrix(int *mat, int sz)
{
	int i, j;
	for (i = 0; i < sz; i++)
	{
		for (j = 0; j < sz; j++)
		{
			printf("%d\t", mat[i * sz + j]);
		}
		printf("\n");
	}
}

int main()
{
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	initialize_gpu_parameters(devProp);
	initialize_timing_and_randomness();

	printf("Enter matrix size:");
	int mat_sz;
	scanf("%d", &mat_sz);
	getchar();

	printf("Would you like to see the random matrices and the resultant matrix?(YES - 1, NO - 0):");
	int is_debug_mode;
	scanf("%d", &is_debug_mode);
	getchar();

	const int MATRIX_SIZE = mat_sz;
	int *host_mat1, *host_mat2, *host_mat3, *device_mat1, *device_mat2, *device_mat3;
	int no_blks, no_threads;
	int i = 0, j = 0;

	host_mat1 = (int *)malloc(MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
	for (i = 0; i < MATRIX_SIZE; i++)
	{
		for (j = 0; j < MATRIX_SIZE; j++)
		{
			host_mat1[i * MATRIX_SIZE + j] = rand() % 10;
		}
	}

	host_mat2 = (int *)malloc(MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
	for (i = 0; i < MATRIX_SIZE; i++)
	{
		for (j = 0; j < MATRIX_SIZE; j++)
		{
			host_mat2[i * MATRIX_SIZE + j] = rand() % 10;
		}
	}

	host_mat3 = (int *)malloc(MATRIX_SIZE * MATRIX_SIZE * sizeof(int *));

	if (is_debug_mode)
	{
		printf("Matrix 1:\n");
		display_matrix(host_mat1, MATRIX_SIZE);
		printf("Matrix 2:\n");
		display_matrix(host_mat2, MATRIX_SIZE);
	}

	QueryPerformanceCounter(&start);

	hipMalloc((void **)&device_mat1, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
	hipMalloc((void **)&device_mat2, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
	hipMalloc((void **)&device_mat3, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));

	hipMemcpy(device_mat1, host_mat1, MATRIX_SIZE * MATRIX_SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_mat2, host_mat2, MATRIX_SIZE * MATRIX_SIZE * sizeof(int), hipMemcpyHostToDevice);

	const int total_elements = MATRIX_SIZE * MATRIX_SIZE;
	if (total_elements <= NUMBER_OF_THREADS_PER_SM)
	{
		no_blks = 1;
		no_threads = total_elements;
	}
	else
	{
		no_threads = NUMBER_OF_THREADS_PER_SM;
		no_blks = total_elements / NUMBER_OF_THREADS_PER_SM + 1;
	}

	matmul << <dim3(no_blks), dim3(no_threads) >> >(device_mat1, device_mat2, device_mat3, MATRIX_SIZE, NUMBER_OF_THREADS_PER_SM);

	hipMemcpy(host_mat3, device_mat3, MATRIX_SIZE * MATRIX_SIZE * sizeof(int), hipMemcpyDeviceToHost);

	QueryPerformanceCounter(&end);

	if (is_debug_mode)
	{
		printf("Result:\n");
		display_matrix(host_mat3, MATRIX_SIZE);
	}

	printf("Time taken:\n");
	long long time_taken = (end.QuadPart - start.QuadPart) / (freq.QuadPart / 1000);
	printf("%lld ms", time_taken);

	free(host_mat1);
	free(host_mat2);
	free(host_mat3);

	hipFree(device_mat1);
	hipFree(device_mat2);
	hipFree(device_mat3);

	getchar();

	return 0;
}