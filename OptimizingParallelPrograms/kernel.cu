#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <stdint.h>
#include <windows.h>
#include <pthread.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""

int MATRIX_SIZE;
int NUMBER_OF_SM, NUMBER_OF_THREADS_PER_SM;
int NUMBER_OF_CPU_BLOCKS;
int no_blks, no_threads;
int is_debug_mode, is_gpu_used;
int *mat1, *mat2, *pthread_mat3, *cuda_mat3, *final_result_mat3;
int *device_mat1, *device_mat2, *device_mat3;

LARGE_INTEGER start, end;
LARGE_INTEGER freq;

void* pthread_mat_mul(void *arg)
{
	int i, j, k;
	i = *(int *)arg;
	j = *((int *)(arg)+1);
	for (k = 0; k < MATRIX_SIZE; k++)
	{
		pthread_mat3[i * MATRIX_SIZE + j] += mat1[i * MATRIX_SIZE + k] * mat2[k * MATRIX_SIZE + j];
	}

	return (void *)0;
}

void calculate_results_pthreads()
{
	const int arg_sz = MATRIX_SIZE * MATRIX_SIZE;
	int i = 0, j = 0, k = 0;

	pthread_t *threads = (pthread_t *)malloc(MATRIX_SIZE * MATRIX_SIZE * sizeof(pthread_t));
	int *thread_args = (int *)malloc(arg_sz * 2 * sizeof(int));

	if (!is_gpu_used)
	{
		for (i = 0; i < MATRIX_SIZE; i++)
		{
			for (j = 0; j < MATRIX_SIZE; j++, k += 2)
			{
				thread_args[k] = i;
				thread_args[k + 1] = j;
				pthread_create(&threads[i * MATRIX_SIZE + j], NULL, pthread_mat_mul, &thread_args[k]);
			}
		}

		for (i = 0; i < MATRIX_SIZE; i++)
		{
			for (j = 0; j < MATRIX_SIZE; j++)
			{
				pthread_join(threads[i * MATRIX_SIZE + j], NULL);
			}
		}
	}
	else
	{
		for (i = 0; i < MATRIX_SIZE; i++)
		{
			for (j = 0; j < MATRIX_SIZE; j++, k += 2)
			{
				int ele = i * MATRIX_SIZE + j;
				int blk = ele / no_threads;
				if (blk >= no_blks)
				{
					thread_args[k] = i;
					thread_args[k + 1] = j;
					pthread_create(&threads[i * MATRIX_SIZE + j], NULL, pthread_mat_mul, &thread_args[k]);
				}
			}
		}

		for (i = 0; i < MATRIX_SIZE; i++)
		{
			for (j = 0; j < MATRIX_SIZE; j++)
			{
				int ele = i * MATRIX_SIZE + j;
				int blk = ele / no_threads;
				if (blk >= no_blks)
				{
					pthread_join(threads[i * MATRIX_SIZE + j], NULL);
				}
			}
		}
	}
}

__global__ void matmul(int *device_mat1, int *device_mat2, int *device_mat3, int sz, int max_thread)
{
	int row, col, ele, i;
	int temp = 0;
	ele = blockIdx.x * max_thread + threadIdx.x + 1;
	row = (ele - 1) / sz;
	if (row >= sz)
		return;
	col = (ele - 1) % sz;
	for (i = 0; i < sz; i++)
	{
		temp += device_mat1[row * sz + i] * device_mat2[i * sz + col];
	}
	device_mat3[row * sz + col] = temp;
}

void calculate_results_cuda()
{
	int i = 0, j = 0;

	hipMalloc((void **)&device_mat1, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
	hipMalloc((void **)&device_mat2, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
	hipMalloc((void **)&device_mat3, MATRIX_SIZE * MATRIX_SIZE * sizeof(int));

	hipMemcpy(device_mat1, mat1, MATRIX_SIZE * MATRIX_SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_mat2, mat2, MATRIX_SIZE * MATRIX_SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(device_mat3, cuda_mat3, MATRIX_SIZE * MATRIX_SIZE * sizeof(int), hipMemcpyHostToDevice);

	const int total_elements = MATRIX_SIZE * MATRIX_SIZE;
	no_threads = NUMBER_OF_THREADS_PER_SM;
	no_blks = (total_elements / NUMBER_OF_THREADS_PER_SM + 1) - NUMBER_OF_CPU_BLOCKS;

	matmul << <dim3(no_blks), dim3(no_threads) >> >(device_mat1, device_mat2, device_mat3, MATRIX_SIZE, NUMBER_OF_THREADS_PER_SM);

	hipMemcpy(cuda_mat3, device_mat3, MATRIX_SIZE * MATRIX_SIZE * sizeof(int), hipMemcpyDeviceToHost);
}

void calculate_results()
{
	if (!is_gpu_used)
	{
		calculate_results_pthreads();
		QueryPerformanceCounter(&end);
	}
	else
	{
		calculate_results_cuda();
		calculate_results_pthreads();
		QueryPerformanceCounter(&end);
		for (int i = 0; i < MATRIX_SIZE; i++)
		{
			for (int j = 0; j < MATRIX_SIZE; j++)
			{
				if (cuda_mat3[i * MATRIX_SIZE + j] != 0)
					final_result_mat3[i * MATRIX_SIZE + j] = cuda_mat3[i * MATRIX_SIZE + j];
				else
					final_result_mat3[i * MATRIX_SIZE + j] = pthread_mat3[i * MATRIX_SIZE + j];
			}
		}
	}
}

inline int _ConvertSMVer2Cores(int major, int minor)
{
	// Defines for GPU Architecture types (using the SM version to determine the # of cores per SM
	typedef struct
	{
		int SM; // 0xMm (hexidecimal notation), M = SM Major version, and m = SM minor version
		int Cores;
	} sSMtoCores;

	sSMtoCores nGpuArchCoresPerSM[] =
	{
		{ 0x20, 32 }, // Fermi Generation (SM 2.0) GF100 class
		{ 0x21, 48 }, // Fermi Generation (SM 2.1) GF10x class
		{ 0x30, 192 }, // Kepler Generation (SM 3.0) GK10x class
		{ 0x32, 192 }, // Kepler Generation (SM 3.2) GK10x class
		{ 0x35, 192 }, // Kepler Generation (SM 3.5) GK11x class
		{ 0x37, 192 }, // Kepler Generation (SM 3.7) GK21x class
		{ 0x50, 128 }, // Maxwell Generation (SM 5.0) GM10x class
		{ 0x52, 128 }, // Maxwell Generation (SM 5.2) GM20x class
		{ 0x53, 128 }, // Maxwell Generation (SM 5.3) GM20x class
		{ 0x60, 64 }, // Pascal Generation (SM 6.0) GP100 class
		{ 0x61, 128 }, // Pascal Generation (SM 6.1) GP10x class
		{ 0x62, 128 }, // Pascal Generation (SM 6.2) GP10x class
		{ -1, -1 }
	};

	int index = 0;

	while (nGpuArchCoresPerSM[index].SM != -1)
	{
		if (nGpuArchCoresPerSM[index].SM == ((major << 4) + minor))
		{
			return nGpuArchCoresPerSM[index].Cores;
		}

		index++;
	}

	// If we don't find the values, we default use the previous one to run properly
	printf("MapSMtoCores for SM %d.%d is undefined.  Default to use %d Cores/SM\n", major, minor, nGpuArchCoresPerSM[index - 1].Cores);
	return nGpuArchCoresPerSM[index - 1].Cores;
}

void initialize_gpu_parameters()
{
	hipDeviceProp_t devProp;
	hipGetDeviceProperties(&devProp, 0);
	NUMBER_OF_SM = devProp.multiProcessorCount;
	NUMBER_OF_THREADS_PER_SM = _ConvertSMVer2Cores(devProp.major, devProp.minor);
}

void initialize_timing_and_randomness()
{
	srand(time(NULL));
	if (QueryPerformanceFrequency(&freq) == 0)
	{
		printf("Your machine not support high resolution performance counter\n");
		return;
	}
}

void check_if_gpu_is_needed()
{
	if (NUMBER_OF_CPU_BLOCKS * NUMBER_OF_THREADS_PER_SM >= MATRIX_SIZE * MATRIX_SIZE)
	{
		is_gpu_used = 0;
		printf("GPU not needed!\n");
	}
	else
	{
		is_gpu_used = 1;
		printf("Using GPU!\n");
	}
}

void display_matrix(int *mat, int sz)
{
	int i, j;
	for (i = 0; i < sz; i++)
	{
		for (j = 0; j < sz; j++)
		{
			printf("%d\t", mat[i * sz + j]);
		}
		printf("\n");
	}
}

void get_user_input()
{
	printf("Enter matrix size:");
	int mat_sz;
	scanf("%d", &MATRIX_SIZE);
	getchar();

	printf("Enter CPU/GPU factor:");
	scanf("%d", &NUMBER_OF_CPU_BLOCKS);
	getchar();

	printf("Would you like to see the random matrices and the resultant matrix?(YES - 1, NO - 0):");
	scanf("%d", &is_debug_mode);
	getchar();
}

void print_results()
{
	if (is_debug_mode)
	{
		printf("Result:\n");
		if (is_gpu_used)
			display_matrix(final_result_mat3, MATRIX_SIZE);
		else
			display_matrix(pthread_mat3, MATRIX_SIZE);
	}

	printf("Time taken:\n");
	long long time_taken = (end.QuadPart - start.QuadPart) / (freq.QuadPart / 1000);
	printf("%lld ms", time_taken);
}

void initialize_random_matrices()
{
	int i = 0, j = 0;

	mat1 = (int *)malloc(MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
	for (i = 0; i < MATRIX_SIZE; i++)
	{
		for (j = 0; j < MATRIX_SIZE; j++)
		{
			mat1[i * MATRIX_SIZE + j] = (rand() % 99) + 1;
		}
	}

	mat2 = (int *)malloc(MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
	for (i = 0; i < MATRIX_SIZE; i++)
	{
		for (j = 0; j < MATRIX_SIZE; j++)
		{
			mat2[i * MATRIX_SIZE + j] = (rand() % 99) + 1;
		}
	}

	if (is_debug_mode)
	{
		printf("Matrix 1:\n");
		display_matrix(mat1, MATRIX_SIZE);
		printf("Matrix 2:\n");
		display_matrix(mat2, MATRIX_SIZE);
	}

	pthread_mat3 = (int *)malloc(MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
	for (i = 0; i < MATRIX_SIZE; i++)
	{
		for (j = 0; j < MATRIX_SIZE; j++)
		{
			pthread_mat3[i * MATRIX_SIZE + j] = 0;
		}
	}

	cuda_mat3 = (int *)malloc(MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
	for (i = 0; i < MATRIX_SIZE; i++)
	{
		for (j = 0; j < MATRIX_SIZE; j++)
		{
			cuda_mat3[i * MATRIX_SIZE + j] = 0;
		}
	}

	final_result_mat3 = (int *)malloc(MATRIX_SIZE * MATRIX_SIZE * sizeof(int));
}

void free_matrices()
{
	free(mat1);
	free(mat2);
	free(cuda_mat3);
	free(pthread_mat3);
	free(final_result_mat3);

	if (is_gpu_used)
	{
		hipFree(device_mat1);
		hipFree(device_mat2);
		hipFree(device_mat3);
	}
}

int main()
{
	initialize_gpu_parameters();
	get_user_input();
	check_if_gpu_is_needed();
	initialize_timing_and_randomness();
	initialize_random_matrices();

	QueryPerformanceCounter(&start);
	calculate_results();
//	QueryPerformanceCounter(&end);

	print_results();
	free_matrices();

	getchar();
	return 0;
}